#include "hip/hip_runtime.h"
#ifndef SUM_H
#define SUM_H

#include "common.h"

/*
 This version adds multiple elements per thread sequentially.  This reduces the overall
 cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
 (Brent's Theorem optimization)

 Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
 In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
 If blockSize > 32, allocate blockSize*sizeof(T) bytes.
 */
template<class T, unsigned int blockSize, bool nIsPow2>
__global__ void sum(T *g_idata, T *g_odata, unsigned int n) {
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n) {
        mySum += g_idata[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i + blockSize];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256)) {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) && (tid < 128)) {
        sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

    __syncthreads();

    if ((blockSize >= 128) && (tid < 64)) {
        sdata[tid] = mySum = mySum + sdata[tid + 64];
    }

    __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >= 64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2)
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockSize >= 64) && (tid < 32)) {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >= 32) && (tid < 16)) {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >= 16) && (tid < 8)) {
        sdata[tid] = mySum = mySum + sdata[tid + 8];
    }

    __syncthreads();

    if ((blockSize >= 8) && (tid < 4)) {
        sdata[tid] = mySum = mySum + sdata[tid + 4];
    }

    __syncthreads();

    if ((blockSize >= 4) && (tid < 2)) {
        sdata[tid] = mySum = mySum + sdata[tid + 2];
    }

    __syncthreads();

    if ((blockSize >= 2) && (tid < 1)) {
        sdata[tid] = mySum = mySum + sdata[tid + 1];
    }

    __syncthreads();
#endif

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = mySum;
}

template __global__ void sum<int, SUM_BLOCK_SIZE, false>(int *g_idata, int *g_odata, unsigned int n);

#endif
