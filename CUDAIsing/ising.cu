#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_int_distribution.hpp>
#include <ctime>
#include <iostream>
#include <stdio.h>

using namespace std;
using namespace boost::random;

static const int SEED = 5;

static const int L = 32;
static const int L3 = L * L * L;
static const int BLOCKS_X = 4;
static const int BLOCK_SIZE = L / BLOCKS_X;

static const int SUM_NUM_BLOCKS = 32;
static const int SUM_BLOCK_SIZE = L3 / SUM_NUM_BLOCKS / 2;
//static const double B = 0;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		cerr << "Error " << _m_cudaStat << ": "                             \
                << hipGetErrorString(_m_cudaStat) << " at line "	        \
				<< __LINE__ << " in file " << __FILE__ << endl;			    \
		exit(1);															\
	}																		\
}

__device__ dim3 getIndex() {
    dim3 index;
    index.x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    index.y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    index.z = blockIdx.z * BLOCK_SIZE + threadIdx.z;
    //printf("(%d,%d,%d) (%d,%d,%d)\n",blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
    return index;
}

__device__ unsigned int getTid(dim3 index) {
    return index.x * L * L + index.y * L + index.z;

}

__device__ int randomSpin(hiprandState * const rngStates, unsigned int tid) {
    int rnd = hiprand(&rngStates[tid]);
    //printf("%f\n", rnd);
    int binary = (rnd >> (tid & 31)) & 1;
    return 2 * binary - 1;
}

__global__ void initRNG(hiprandState * const rngStates,
        const unsigned int seed) {
    unsigned int tid = blockIdx.x * SUM_BLOCK_SIZE + threadIdx.x;
    if (tid < L3) {
        hiprand_init(seed, tid, 0, &rngStates[tid]);
    }
}

__global__ void fillMatrix(int* S, hiprandState * const rngStates) {
    unsigned int tid = blockIdx.x * SUM_BLOCK_SIZE + threadIdx.x;
    if (tid < L3) {
        //skipahead(100, &rngStates[tid]);
        S[tid] = randomSpin(rngStates, tid);
    }
}

template<char dir, int skip> __device__ int neigh(dim3 index) {
    dim3 result = index;
    if (dir == 'x')
        result.x = (result.x + skip) % L;
    if (dir == 'y')
        result.y = (result.y + skip) % L;
    if (dir == 'z')
        result.z = (result.z + skip) % L;
    return getTid(result);
}

__device__ int energy(int* S, dim3 index, int tid) {
    int nEnergy = S[neigh<'x', 1>(index)] + S[neigh<'x', -1>(index)] + S[neigh<'y', 1>(index)]
            + S[neigh<'y', -1>(index)] + S[neigh<'z', 1>(index)] + S[neigh<'z', -1>(index)];
    return -S[tid] * nEnergy;
}

__device__ void tryInvert(int* S, dim3 index, float beta,
        hiprandState * const rngStates) {
    if (index.x < L && index.y < L && index.z < L) {
        unsigned int tid = getTid(index);
        int dE = -2 * energy(S, index, tid);
        if (dE < 0 || hiprand_uniform(&(rngStates[tid])) < __expf(-beta * dE))
            S[tid] = -S[tid];
    }
}

template<bool second> __global__ void generateNext(int* S, float beta, hiprandState * const rngStates) {
    dim3 index = getIndex();
    int shifting = (index.x ^ index.y) & 1;
    //printf("(%d,%d,%d) shifting %d second %d\n",index.x, index.y, index.z, shifting, second);
    if (second)
        shifting = 1 - shifting;
    index.z = (2 * index.z) + shifting;
    //printf("(%d,%d,%d)\n",index.x, index.y, index.z);
    tryInvert(S, index, beta, rngStates);
}

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory {
    __device__ inline operator T *() {
        extern __shared__ int __smem[];
        return (T *) __smem;
    }

    __device__ inline operator const T *() const {
        extern __shared__ int __smem[];
        return (T *) __smem;
    }
};

/*
 This version adds multiple elements per thread sequentially.  This reduces the overall
 cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
 (Brent's Theorem optimization)

 Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
 In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
 If blockSize > 32, allocate blockSize*sizeof(T) bytes.
 */
template<class T, unsigned int blockSize, bool nIsPow2>
__global__ void sum(T *g_idata, T *g_odata, unsigned int n) {
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n) {
        mySum += g_idata[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i + blockSize];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256)) {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) && (tid < 128)) {
        sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

    __syncthreads();

    if ((blockSize >= 128) && (tid < 64)) {
        sdata[tid] = mySum = mySum + sdata[tid + 64];
    }

    __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >= 64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2)
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockSize >= 64) && (tid < 32)) {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >= 32) && (tid < 16)) {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >= 16) && (tid < 8)) {
        sdata[tid] = mySum = mySum + sdata[tid + 8];
    }

    __syncthreads();

    if ((blockSize >= 8) && (tid < 4)) {
        sdata[tid] = mySum = mySum + sdata[tid + 4];
    }

    __syncthreads();

    if ((blockSize >= 4) && (tid < 2)) {
        sdata[tid] = mySum = mySum + sdata[tid + 2];
    }

    __syncthreads();

    if ((blockSize >= 2) && (tid < 1)) {
        sdata[tid] = mySum = mySum + sdata[tid + 1];
    }

    __syncthreads();
#endif

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = mySum;
}

__global__ void print(int* S) {
    for (int i = 0; i < L; ++i)
        for (int j = 0; j < L; ++j)
            for (int k = 0; k < L; ++k) {
                dim3 index(i, j, k);
                printf("%d,%d,%d: %d\n", i, j, k, S[getTid(index)]);
            }
}

__global__ void totalEnergy(int* S) {
    int e = 0;
    for (int i = 0; i < L; ++i)
        for (int j = 0; j < L; ++j)
            for (int k = 0; k < L; ++k) {
                dim3 index(i, j, k);
                e += energy(S, index, getTid(index));
            }
    printf("Total energy= %d\n", e);
}

class Configuration {
public:
    Configuration(float T, int seed = time(0)) :
            T(T) {
        beta = 1 / T;

        hipExtent extent = make_hipExtent(L * sizeof(int), L, L);

        hipPitchedPtr ptrS;
        CUDA_CHECK_RETURN(hipMalloc3D(&ptrS, extent));
        this->ptrS = (int*) ptrS.ptr;

        blocks = dim3(BLOCKS_X, BLOCKS_X, BLOCKS_X/2);
        threads = dim3(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);

        //cout<< blocks.x <<blocks.y << blocks.z <<endl;
        //cout << threads.x << threads.y << threads.z <<endl;

        CUDA_CHECK_RETURN(hipMalloc(&rngStates, L3 * sizeof(hiprandState)));

        initRNG<<<SUM_NUM_BLOCKS * 2, SUM_BLOCK_SIZE>>>(rngStates, seed);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
        fillMatrix<<<SUM_NUM_BLOCKS * 2, SUM_BLOCK_SIZE>>>(this->ptrS,
                rngStates);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());

        CUDA_CHECK_RETURN(
                hipMalloc(&deviceSumPtr, sizeof(int) * SUM_NUM_BLOCKS));
    }

    ~Configuration() {
        CUDA_CHECK_RETURN(hipGetLastError());

        CUDA_CHECK_RETURN(hipFree(rngStates));
        CUDA_CHECK_RETURN(hipFree(ptrS));
        CUDA_CHECK_RETURN(hipFree(deviceSumPtr));
        CUDA_CHECK_RETURN(hipDeviceReset());
    }

    void nextConfig() {
        generateNext<false><<<blocks, threads>>>(ptrS, beta, rngStates);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
        generateNext<true><<<blocks, threads>>>(ptrS, beta, rngStates);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
    }

    double getMagnet() {
        sum<int, SUM_BLOCK_SIZE, true> <<<SUM_NUM_BLOCKS, SUM_BLOCK_SIZE,
                SUM_BLOCK_SIZE * sizeof(int)>>>((int *) ptrS, deviceSumPtr, L3);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
        CUDA_CHECK_RETURN(
                hipMemcpy(hostSumPtr, deviceSumPtr,
                        sizeof(int) * SUM_NUM_BLOCKS, hipMemcpyDeviceToHost));
        CUDA_CHECK_RETURN(hipGetLastError());
        double result = 0;
        for (int i = 0; i < SUM_NUM_BLOCKS; ++i) {
            result += hostSumPtr[i];
        }
        return result;
    }

    void printMatrix(int i) {
        cout << "iteration " << i << endl;
        print<<<1, 1>>>(ptrS);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
    }

    void printEnergy(int i) {
        cout << "iteration " << i << ": ";
        totalEnergy<<<1, 1>>>(ptrS);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
    }

private:
    mt19937 gen;
    uniform_int_distribution<> distrib;
    const float T;
    float beta;

    hiprandState *rngStates;
    dim3 blocks;
    dim3 threads;
    int* ptrS;
    int* deviceSumPtr;
    int hostSumPtr[SUM_NUM_BLOCKS];
};

int main(int argc, char** argv) {
    double T = 0.1;
    unsigned int N = 10;
    if (argc >= 2)
        T = atof(argv[1]);
    if (argc >= 3)
        N = atoi(argv[2]);
    Configuration S(T, time(0));
    double sum = 0;
    for (int i = 0; i < N; i++) {
        S.nextConfig();
        //S.printMatrix(i);
        double magnet = S.getMagnet();
        sum += magnet;
        //S.printEnergy(i);
        //cout << magnet << endl;
    }
    cout << sum / N << endl;
}
