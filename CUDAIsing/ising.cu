#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <boost/random/mersenne_twister.hpp>
#include <boost/random/uniform_int_distribution.hpp>
#include <ctime>
#include <iostream>
#include <stdio.h>

using namespace std;
using namespace boost::random;

static const int SEED = 5;

static const int L = 32;
static const int L3 = L * L * L;
static const int BLOCKS_X = 4;
static const int BLOCK_SIZE = L / BLOCKS_X;

static const int SUM_NUM_BLOCKS = 32;
static const int SUM_BLOCK_SIZE = L3 / SUM_NUM_BLOCKS / 2;
//static const double B = 0;

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		cerr << "Error " << _m_cudaStat << ": "                             \
                << hipGetErrorString(_m_cudaStat) << " at line "	        \
				<< __LINE__ << " in file " << __FILE__ << endl;			    \
		exit(1);															\
	}																		\
}

__device__ dim3 getIndex() {
    dim3 index;
    index.x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    index.y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    index.z = blockIdx.z * BLOCK_SIZE + threadIdx.z;
    return index;
}

__device__ unsigned int getTid(dim3 index) {
    return index.x * L * L + index.y * L + index.z;

}

__device__ float rand(hiprandState * const rngStates, unsigned int tid) {
    return hiprand_uniform(&rngStates[tid]);
}

__device__ int randomSpin(hiprandState * const rngStates, unsigned int tid) {
    int binary = (int) (1 - rand(rngStates, tid) + 0.5);
    return 2 * binary - 1;
}

__global__ void initRNG(hipPitchedPtr Sptr, hiprandState * const rngStates,
        const unsigned int seed) {
    unsigned int tid = blockIdx.x * SUM_BLOCK_SIZE + threadIdx.x;
    if (tid < L3) {
        int* S = (int *) Sptr.ptr;
        hiprand_init(seed, tid, 0, &rngStates[tid]);
        skipahead(100, &rngStates[tid]);

        S[tid] = randomSpin(rngStates, tid);
    }

}

__device__ void tryInvert(int* S, dim3 index, float beta,
        hiprandState * const rngStates) {
    if (index.x < L && index.y < L && index.z < L) {
        //left
        index.z = (index.z + 1) % L;
        int neigh = S[getTid(index)];
        //right
        index.z = (index.z + L - 2) % L;
        neigh += S[getTid(index)];
        //up
        index.z = (index.z + 1) % L;
        index.y = (index.y + 1) % L;
        neigh += S[getTid(index)];
        //down
        index.y = (index.y + L - 2) % L;
        neigh += S[getTid(index)];
        //forward
        index.y = (index.y + 1) % L;
        index.x = (index.x + 1) % L;
        neigh += S[getTid(index)];
        //backward
        index.x = (index.x + L - 2) % L;
        neigh += S[getTid(index)];
        index.x = (index.x + 1) % L;
        //energy
        unsigned int tid = getTid(index);
        int dE = -2 * S[tid] * neigh;
        if (dE < 0 || rand(rngStates, tid) < __expf(-beta * dE))
            S[tid] *= -1;
    }
}

__global__ void generateNext(hipPitchedPtr Sptr, float beta,
        hiprandState * const rngStates) {
    int* S = (int *) Sptr.ptr;
    dim3 index = getIndex();
    index.z = 2 * index.z + (index.x & 1) ^ (index.y & 1);
    tryInvert(S, index, beta, rngStates);
    index.z++;
    tryInvert(S, index, beta, rngStates);
}

__global__ void sum(hipPitchedPtr Sptr, int* output) {
    int* S = (int *) Sptr.ptr;
    //@@ Load a segment of the S vector into shared memory
    __shared__ int partialSum[SUM_BLOCK_SIZE * 2];
    unsigned int i = threadIdx.x, start = 2 * blockIdx.x * SUM_BLOCK_SIZE;
    if (start + i < L3)
        partialSum[i] = S[start + i];
    else
        partialSum[i] = 0;
    if (start + SUM_BLOCK_SIZE + i < L3)
        partialSum[SUM_BLOCK_SIZE + i] = S[start + SUM_BLOCK_SIZE + i];
    else
        partialSum[SUM_BLOCK_SIZE + i] = 0;
    //@@ Traverse the reduction tree
    for (unsigned int stride = SUM_BLOCK_SIZE; stride >= 1; stride >>= 1) {
        __syncthreads();
        if (i < stride)
            partialSum[i] += partialSum[i + stride];
    }
    //@@ Write the computed sum of the block to the output vector at the
    //@@ correct index
    if (i == 0)
        output[blockIdx.x] = partialSum[0];
}

__global__ void print(hipPitchedPtr Sptr) {
    int* S = (int *) Sptr.ptr;
    for (int i = 0; i < L3; ++i) {
        printf("%d: %d\n", i, S[i]);
    }
}

class Configuration {
public:
    Configuration(float T, int seed = time(0)) :
            T(T) {
        beta = 1 / T;

        hipExtent extent = make_hipExtent(L * sizeof(int), L, L);
        CUDA_CHECK_RETURN(hipMalloc3D(&ptr, extent));

        blocks = dim3(BLOCKS_X, BLOCKS_X, BLOCKS_X);
        threads.x = threads.y = BLOCK_SIZE;
        threads.z = BLOCK_SIZE / 2;

        CUDA_CHECK_RETURN(hipMalloc(&rngStates, L3 * sizeof(hiprandState)));
        initRNG<<<SUM_NUM_BLOCKS, SUM_BLOCK_SIZE>>>(ptr, rngStates, seed);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());

        CUDA_CHECK_RETURN(
                hipMalloc(&deviceSumPtr, sizeof(int) * SUM_NUM_BLOCKS));
    }

    ~Configuration() {
        CUDA_CHECK_RETURN(hipGetLastError());

        CUDA_CHECK_RETURN(hipFree(rngStates));
        CUDA_CHECK_RETURN(hipFree(ptr.ptr));
        CUDA_CHECK_RETURN(hipDeviceReset());
    }

    void nextConfig() {
        generateNext<<<blocks, threads>>>(ptr, beta, rngStates);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
    }

    double getMagnet() {
        sum<<<SUM_NUM_BLOCKS, SUM_BLOCK_SIZE>>>(ptr, deviceSumPtr);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
        CUDA_CHECK_RETURN(
                hipMemcpy(hostSumPtr, deviceSumPtr,
                        sizeof(int) * SUM_NUM_BLOCKS, hipMemcpyDeviceToHost));
        CUDA_CHECK_RETURN(hipGetLastError());
        double result = 0;
        for (int i = 0; i < SUM_NUM_BLOCKS; ++i) {
            result += hostSumPtr[i];
        }
        return result;
    }

private:
    mt19937 gen;
    uniform_int_distribution<> spindist;
    const float T;
    int matrix[L][L][L];
    float beta;

    hiprandState *rngStates;
    dim3 blocks;
    dim3 threads;
    hipPitchedPtr ptr;
    int* deviceSumPtr;
    int hostSumPtr[SUM_NUM_BLOCKS];
};

int main(int argc, char** argv) {
    double T = 0;
    unsigned int N = 2;
    if (argc >= 2)
        T = atoi(argv[1]);
    if (argc >= 3)
        N = atoi(argv[2]);
    Configuration S(T, SEED);
    double sum = 0;
    for (int i = 0; i < N; i++) {
        S.nextConfig();
        sum += S.getMagnet();
    }
    cout << sum / N << endl;
}
