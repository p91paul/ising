#include "hip/hip_runtime.h"
#include "common.h"
#include "random.h"


__device__ int randomSpin(hiprandState * const rngStates, unsigned int tid) {
    unsigned int rnd = hiprand(&rngStates[tid]);
    //printf("%f\n", rnd);
    int binary = (rnd >> ((rnd ^ tid) & 31)) & 1;
    return (binary << 1) - 1;
}

__global__ void initRNG(hiprandState * const rngStates,
        const unsigned int seed) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < L3) {
        hiprand_init(seed, tid, 0, &rngStates[tid]);
    }
}

__global__ void fillMatrix(int* S, hiprandState * const rngStates) {
    unsigned int tid = blockIdx.x * SUM_BLOCK_SIZE + threadIdx.x;
    if (tid < L3) {
        //skipahead(100, &rngStates[tid]);
        S[tid] = randomSpin(rngStates, tid);
    }
}
