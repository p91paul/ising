#include "hip/hip_runtime.h"
#include "common.h"
#include "random.h"


__device__ int randomSpin(hiprandState * const rngStates, unsigned int tid) {
    int rnd = hiprand(&rngStates[tid]);
    //printf("%f\n", rnd);
    int binary = (rnd >> (tid & 31)) & 1;
    return 2 * binary - 1;
}

__global__ void initRNG(hiprandState * const rngStates,
        const unsigned int seed) {
    unsigned int tid = blockIdx.x * SUM_BLOCK_SIZE + threadIdx.x;
    if (tid < L3) {
        hiprand_init(seed, tid, 0, &rngStates[tid]);
    }
}

__global__ void fillMatrix(int* S, hiprandState * const rngStates) {
    unsigned int tid = blockIdx.x * SUM_BLOCK_SIZE + threadIdx.x;
    if (tid < L3) {
        //skipahead(100, &rngStates[tid]);
        S[tid] = randomSpin(rngStates, tid);
    }
}
