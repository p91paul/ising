#include <ctime>
#include <iostream>

using namespace std;
//static const double B = 0;

#include "common.h"
#include "sum.h"
#include "random.h"
#include "ising.h"

class Configuration {
public:
    Configuration(float T, int seed = time(0)) :
            T(T) {
        beta = 1 / T;

        hipExtent extent = make_hipExtent(L * sizeof(int), L, L);

        hipPitchedPtr ptrS;
        CUDA_CHECK_RETURN(hipMalloc3D(&ptrS, extent));
        this->ptrS = (int*) ptrS.ptr;

        blocks = dim3(BLOCKS_XY, BLOCKS_XY, BLOCKS_Z);
        threads = dim3(BLOCK_SIZE_XY, BLOCK_SIZE_XY, BLOCK_SIZE_Z);

        //cout<< blocks.x <<blocks.y << blocks.z <<endl;
        //cout << threads.x << threads.y << threads.z <<endl;

        CUDA_CHECK_RETURN(hipMalloc(&rngStates, L3 * sizeof(hiprandState)));

        initRNG<<<SUM_NUM_BLOCKS * 2, SUM_BLOCK_SIZE>>>(rngStates, seed);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
        fillMatrix<<<SUM_NUM_BLOCKS * 2, SUM_BLOCK_SIZE>>>(this->ptrS,
                rngStates);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());

        CUDA_CHECK_RETURN(
                hipMalloc(&deviceSumPtr, sizeof(int) * SUM_NUM_BLOCKS));
    }

    ~Configuration() {
        CUDA_CHECK_RETURN(hipGetLastError());

        CUDA_CHECK_RETURN(hipFree(rngStates));
        CUDA_CHECK_RETURN(hipFree(ptrS));
        CUDA_CHECK_RETURN(hipFree(deviceSumPtr));
        CUDA_CHECK_RETURN(hipDeviceReset());
    }

    void nextConfig() {
        generateNext<false><<<blocks, threads>>>(ptrS, beta, rngStates);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
        generateNext<true><<<blocks, threads>>>(ptrS, beta, rngStates);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
    }

    double getMagnet() {
        sum<int, SUM_BLOCK_SIZE, false> <<<SUM_NUM_BLOCKS, SUM_BLOCK_SIZE,
                SUM_BLOCK_SIZE * sizeof(int)>>>((int *) ptrS, deviceSumPtr, L3);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
        CUDA_CHECK_RETURN(
                hipMemcpy(hostSumPtr, deviceSumPtr,
                        sizeof(int) * SUM_NUM_BLOCKS, hipMemcpyDeviceToHost));
        CUDA_CHECK_RETURN(hipGetLastError());
        double result = 0;
        for (int i = 0; i < SUM_NUM_BLOCKS; ++i) {
            result += hostSumPtr[i];
        }
        return result;
    }

    void printMatrix(int i) {
        cout << "iteration " << i << endl;
        print<<<1, 1>>>(ptrS);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
    }

    void printEnergy(int i) {
        cout << "iteration " << i << ": ";
        totalEnergy<<<1, 1>>>(ptrS);
        CUDA_CHECK_RETURN(hipDeviceSynchronize()); // Wait for the GPU launched work to complete
        CUDA_CHECK_RETURN(hipGetLastError());
    }

private:
    const float T;
    float beta;

    hiprandState *rngStates;
    dim3 blocks;
    dim3 threads;
    int* ptrS;
    int* deviceSumPtr;
    int hostSumPtr[SUM_NUM_BLOCKS];
};

int main(int argc, char** argv) {
    double T = 0.1;
    unsigned int N = 10;
    if (argc >= 2)
        T = atof(argv[1]);
    if (argc >= 3)
        N = atoi(argv[2]);
    Configuration S(T, SEED);
    double sum = 0;
    for (int i = 0; i < N; i++) {
        S.nextConfig();
        //S.printMatrix(i);
        double magnet = S.getMagnet();
        sum += magnet;
        //S.printEnergy(i);
        //cout << magnet << endl;
    }
    cout << sum / N << endl;
}
